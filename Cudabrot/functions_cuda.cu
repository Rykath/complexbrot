/*
 *  Created on: Oct 29, 2017
 *      Author: Rykath
 */


#include <hip/hip_runtime.h>
#include <string>
#include <math.h>
#include <stdio.h>

__global__ void sequence_mandel(int* escIter, float* cr, float* ci, int maxIter){
	int pos = blockIdx.x*blockDim.x + threadIdx.x;
	
	float zr = 0;
	float zi = 0;
	float zr2 = 0;	// zr squared
	float zi2 = 0;	// zi squared
	
	for (int iter=0; iter<maxIter; iter++){
		zi = 2*zi*zr + ci[pos]; // !! order is important, zi has to be first
		zr = zr2 - zi2 + cr[pos];
		zr2 = zr*zr;
		zi2 = zi*zi;
		if (zr2 + zi2 > 4.0){	// escaping
			escIter[pos] = iter;
			return;
		}
	}
	escIter[pos] = maxIter;
	return;
}

int* sector_mandel(float cenCr, float cenCi, float widthC, int widthSpl, int iterations){
	int* escIter, * escIter_g;	// dual memory: _g is gpu memory
	float* cr,* cr_g;
	float* ci,* ci_g;
	
	int numSpl = widthSpl*widthSpl;
	
	escIter = (int*)malloc(numSpl*sizeof(int));
	cr = (float*)malloc(numSpl*sizeof(float));
	ci = (float*)malloc(numSpl*sizeof(float));
	
	hipMalloc(&escIter_g, numSpl*sizeof(int));
	hipMalloc(&cr_g, numSpl*sizeof(float));
	hipMalloc(&ci_g, numSpl*sizeof(float));

	float resC = widthC/(float)widthSpl;
	for (int h=0; h<widthSpl; h++){
		for (int w=0; w<widthSpl; w++){
			cr[h*widthSpl+w] = (w-widthSpl/2.0)*resC-cenCr;
			ci[h*widthSpl+w] = (h-widthSpl/2.0)*resC-cenCi;
		}
	}
	
	hipMemcpy(cr_g, cr, numSpl*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(ci_g, ci, numSpl*sizeof(float), hipMemcpyHostToDevice);
    
	sequence_mandel<<<(numSpl+255)/256, 256>>>(escIter_g,cr_g,ci_g,iterations);

	hipMemcpy(escIter, escIter_g, numSpl*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(escIter_g);
	hipFree(cr_g);
	hipFree(ci_g);
	free(cr);
	free(ci);
	
	return escIter;
}

__global__ void sequence_periodicity(int* perIter, float* cr, float* ci, int maxIter, int period, float near){
	int pos = blockIdx.x*blockDim.x + threadIdx.x;
	
	float zr = 0;
	float zi = 0;
	float zr2 = 0;	// zr squared
	float zi2 = 0;	// zi squared
	float pr = 0;	// checking periodicity against
	float pi = 0;
	
	for (int iter=0; iter<maxIter; iter++){
		zi = 2*zi*zr + ci[pos]; // !! order is important, zi has to be first
		zr = zr2 - zi2 + cr[pos];
		zr2 = zr*zr;
		zi2 = zi*zi;
		if (zr2 + zi2 > 4.0){	// escaping
			perIter[pos] = 0;
			return;
		}
		if (iter % period == 0){
			if (abs(pr-zr) < near && abs(pi-zi) < near){
				perIter[pos] = maxIter-iter; // brighter if detected sooner
				return;
			}
			else{
				pr = zr;
				pi = zi;
			}
		}
	}
	perIter[pos] = 0;
	return;
}

int* sector_periodicity(float cenCr, float cenCi, float widthC, int widthSpl, int iterations, int period, float near){
	int* escIter, * escIter_g;	// dual memory: _g is gpu memory
	float* cr,* cr_g;
	float* ci,* ci_g;
	
	int numSpl = widthSpl*widthSpl;
	
	escIter = (int*)malloc(numSpl*sizeof(int));
	cr = (float*)malloc(numSpl*sizeof(float));
	ci = (float*)malloc(numSpl*sizeof(float));
	
	hipMalloc(&escIter_g, numSpl*sizeof(int));
	hipMalloc(&cr_g, numSpl*sizeof(float));
	hipMalloc(&ci_g, numSpl*sizeof(float));

	float resC = widthC/(float)widthSpl;
	for (int h=0; h<widthSpl; h++){
		for (int w=0; w<widthSpl; w++){
			cr[h*widthSpl+w] = (w-widthSpl/2.0)*resC-cenCr;
			ci[h*widthSpl+w] = (h-widthSpl/2.0)*resC-cenCi;
		}
	}
	
	hipMemcpy(cr_g, cr, numSpl*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(ci_g, ci, numSpl*sizeof(float), hipMemcpyHostToDevice);
    
	sequence_periodicity<<<(numSpl+255)/256, 256>>>(escIter_g,cr_g,ci_g,iterations,period,near);

	hipMemcpy(escIter, escIter_g, numSpl*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(escIter_g);
	hipFree(cr_g);
	hipFree(ci_g);
	free(cr);
	free(ci);
	
	return escIter;
}

__global__ void sequence_buddha(int* escIter, float* escPath, float* cr, float* ci, int maxIter){
	int pos = blockIdx.x*blockDim.x + threadIdx.x;
	
	float zr = 0;
	float zi = 0;
	float zr2 = 0;	// zr squared
	float zi2 = 0;	// zi squared
	
	for (int iter=0; iter<maxIter; iter++){
		zi = 2*zi*zr + ci[pos]; // !! order is important, zi has to be first
		zr = zr2 - zi2 + cr[pos];
		zr2 = zr*zr;
		zi2 = zi*zi;
		escPath[pos*maxIter*2 + iter*2] = zr;
		escPath[pos*maxIter*2 + iter*2 + 1] = zi;
		if (zr2 + zi2 > 4.0){	// escaping
			escIter[pos] = iter;
			return;
		}
	}
	escIter[pos] = maxIter;
	return;
}

void sector_buddha(int** retEsc, float** retPath, float cenCr, float cenCi, float widthC, int widthSpl, int iterations){
	int* escIter, * escIter_g;	// dual memory: _g is gpu memory
	float* cr,* cr_g;
	float* ci,* ci_g;
	float* escPath,* escPath_g;
	
	int numSpl = widthSpl*widthSpl;
	
	escIter = (int*)malloc(numSpl*sizeof(int));
	cr = (float*)malloc(numSpl*sizeof(float));
	ci = (float*)malloc(numSpl*sizeof(float));
	escPath = (float*)malloc(numSpl*sizeof(int)*2*iterations);
	
	hipMalloc(&escIter_g, numSpl*sizeof(int));
	hipMalloc(&cr_g, numSpl*sizeof(float));
	hipMalloc(&ci_g, numSpl*sizeof(float));
	hipMalloc(&escPath_g, numSpl*sizeof(float)*2*iterations);

	float resC = widthC/(float)widthSpl;
	for (int h=0; h<widthSpl; h++){
		for (int w=0; w<widthSpl; w++){
			cr[h*widthSpl+w] = (w-widthSpl/2.0)*resC-cenCr;
			ci[h*widthSpl+w] = (h-widthSpl/2.0)*resC-cenCi;
		}
	}
	
	hipMemcpy(cr_g, cr, numSpl*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(ci_g, ci, numSpl*sizeof(float), hipMemcpyHostToDevice);
    
	sequence_buddha<<<(numSpl+255)/256, 256>>>(escIter_g,escPath_g,cr_g,ci_g,iterations);

	hipMemcpy(escIter, escIter_g, numSpl*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(escPath, escPath_g, numSpl*sizeof(float)*2*iterations, hipMemcpyDeviceToHost);
	
	hipFree(escIter_g);
	hipFree(cr_g);
	hipFree(ci_g);
	hipFree(escPath_g);
	
	free(cr);
	free(ci);
	
	*retEsc = escIter;
	*retPath = escPath;
	return;
}
